#include "hip/hip_runtime.h"
#include "../kernels_interface.h"
#include "argmax_kernel.cuh"
#include "tensor/tensor.h"
#include <hip/hip_runtime.h>

namespace kernel {
// 保留原有设备函数（HIP 完全兼容）
__forceinline__ __device__ void warp_reduce_argmax(float& val, size_t& ptr) {
  float tmp_val;
  size_t tmp_ptr;
  unsigned int mask = __ballot_sync(0xFFFFFFFF, true);
  for (unsigned int k = (warpSize >> 1); k > 0; k >>= 1) {
    tmp_val = __shfl_down_sync(mask, val, k, warpSize);
    tmp_ptr = __shfl_down_sync(mask, ptr, k, warpSize);
    if (ptr == SIZE_MAX || tmp_ptr == SIZE_MAX) continue;
    if (tmp_val > val) {
      val = tmp_val;
      ptr = tmp_ptr;
    } else if (tmp_val == val && tmp_ptr < ptr) {
      ptr = tmp_ptr;
    }
  }
}

__forceinline__ __device__ void block_reduce_argmax(float& val, size_t& ptr, float* shared_value,
                                                  size_t* shared_ptr) {
  int lane_id = threadIdx.x % warpSize;
  int warp_id = threadIdx.x / warpSize;

  warp_reduce_argmax(val, ptr);

  __syncthreads();
  if (lane_id == 0) {
    shared_value[warp_id] = val;
    shared_ptr[warp_id] = ptr;
  }

  __syncthreads();
  if (threadIdx.x < blockDim.x / warpSize) {
    val = shared_value[lane_id];
    ptr = shared_ptr[lane_id];
  } else {
    val = 0;
    ptr = SIZE_MAX;
  }

  if (warp_id == 0) {
    warp_reduce_argmax(val, ptr);
  }
}

// 内核函数保持不变
__global__ void argmax_kernel_fp32(const float* input_ptr, size_t size, size_t* output_idx) {
  __shared__ size_t shared_max_ptr[32];
  __shared__ float shared_max_value[32];
  uint32_t tid = threadIdx.x;
  if (tid >= size) {
    return;
  }

  size_t max_index = threadIdx.x;
  float max_value = input_ptr[max_index];
  for (size_t i = tid; i < size; i += blockDim.x) {
    if (input_ptr[i] > max_value) {
      max_index = i;
      max_value = input_ptr[i];
    }
  }

  block_reduce_argmax(max_value, max_index, shared_max_value, shared_max_ptr);
  __syncthreads();
  if (threadIdx.x == 0) {
    *output_idx = max_index;
  }
}

// 替换 CUDA API 为 HIP API
size_t argmax_kernel_hip(const float* input_ptr, size_t size, void* stream) {
  // 替换为 HIP 的内存分配器（需根据项目实际实现调整）
  std::shared_ptr<base::DeviceAllocator> alloc_hip =
      base::HIPDeviceAllocatorFactory::get_instance();  // 假设存在 HIP 分配器
  size_t* index = static_cast<size_t*>(alloc_hip->allocate(sizeof(size_t)));
  size_t output_index = 0;

  if (!stream) {
    argmax_kernel_fp32<<<1, 512>>>(input_ptr, size, index);
    hipMemcpy(&output_index, index, sizeof(size_t), hipMemcpyDeviceToHost);
  } else {
    hipStream_t stream_ = static_cast<hipStream_t>(stream);
    argmax_kernel_fp32<<<1, 512, 0, stream_>>>(input_ptr, size, index);
    hipMemcpyAsync(&output_index, index, sizeof(size_t), hipMemcpyDeviceToHost, stream_);
  }
  return output_index;
}

}  // namespace kernel