#include "hip/hip_runtime.h"
#include "emb_kernel.cuh"
#include <hip/hip_runtime.h>  // 直接使用 HIP 头文件

namespace kernel {

// 内核函数命名改为 _hip 以区分（可选）
__global__ void emb_kernel_hip_fp32(int32_t vocab_size, int32_t token_num, int32_t weight_dim,
                                   const int32_t* input_ptr, const float* weight_ptr,
                                   float* output_ptr) {
  int32_t token_idx = blockIdx.x;
  if (token_idx >= token_num) {
    return;
  }
  int32_t token = input_ptr[token_idx];
  if (token >= vocab_size) {
    return;
  }

  float* output_ptr_start = output_ptr + token_idx * weight_dim;
  const float* weight_ptr_start = weight_ptr + token * weight_dim;

  for (int32_t i = threadIdx.x; i < weight_dim; i += blockDim.x) {
    output_ptr_start[i] = weight_ptr_start[i];
  }
}

void emb_kernel_hip(const tensor::Tensor& input, const tensor::Tensor& weight,
                   const tensor::Tensor& output, int32_t vocab_size, void* stream) {
  tensor::Tensor input_hip;
  if (input.device_type() != base::DeviceType::kDeviceHIP) {  // 修改：kDeviceCUDA -> kDeviceHIP
    input_hip = input.clone();
    input_hip.to_hip();  // 假设有 to_hip() 方法，类似 to_cuda()
  }

  const int32_t input_num = static_cast<int32_t>(input.size());
  const int32_t weight_dim = weight.get_dim(1);
  
  // 检查设备类型是否为 HIP
  CHECK(weight.device_type() == output.device_type());
  CHECK(output.device_type() == base::DeviceType::kDeviceHIP);  // 修改：kDeviceCUDA -> kDeviceHIP

  constexpr int32_t max_seq_len = 512;
  constexpr int32_t thread_num = 128;
  int32_t* in_ptr = input_hip.ptr<int32_t>();
  float* wei_ptr = const_cast<float*>(weight.ptr<float>());
  float* out_ptr = const_cast<float*>(output.ptr<float>());

  if (stream) {
    hipStream_t stream_ = static_cast<hipStream_t>(stream);  // 修改：hipStream_t -> hipStream_t
    emb_kernel_hip_fp32<<<max_seq_len, thread_num, 0, stream_>>>(
        vocab_size, input_num, weight_dim, in_ptr, wei_ptr, out_ptr);
  } else {
    emb_kernel_hip_fp32<<<max_seq_len, thread_num>>>(
        vocab_size, input_num, weight_dim, in_ptr, wei_ptr, out_ptr);
  }
}
}  // namespace kernel